#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cfloat>

#define CUDA_1D_KERNEL_LOOP(i, n)           \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;    \
    i += blockDim.x * gridDim.x)


template <typename scalar_t>
__global__ void align_feature_forward_kernel(const int nthreads,
                                             const scalar_t *data,
                                             const scalar_t *weight,
                                             const int weight_height,
                                             const int weight_width,
                                             const int N,
                                             const int C,
                                             const int Size_Weight,
                                             const int H,
                                             const int W,
                                             scalar_t *output) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int w = i % W;
    const int h = (i / W) % H;
    const int c = (i / (H * W)) % C;
    const int n = (i / (C * H * W));

    int p_h = 0,p_w = 0, p_weight = 0;
    for (int i =0; i < weight_height; i++) {
      for (int j =0; j < weight_width; j++) {
        p_h = (i - weight_height/2) + h;
        p_w = (j - weight_width/2) + w;
        if (p_h >=0 && p_w >=0 && p_h < H && p_w < W) {
          p_weight = i * weight_width + j;
          int data_index = n*C*H*W + c*H*W + h*W + w;
          int weight_index = n*Size_Weight*H*W + p_weight*H*W + h*W +w;
          output[i] += data[data_index] * weight[weight_index];
          }
        }
      }
  }
}


template <typename scalar_t>
__global__ void align_feature_backward_kernel(const int nthreads,
                                              const scalar_t *grad_top,
                                              const scalar_t *data,
                                              const scalar_t *weight,
                                              const int weight_height,
                                              const int weight_width,
                                              const int N,
                                              const int C,
                                              const int Size_Weight,
                                              const int H,
                                              const int W,
                                              scalar_t *grad_data,
                                              scalar_t *grad_weight) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int w = i % W;
    const int h = (i / W) % H;
    const int c = (i / (H * W)) % C;
    const int n = (i / (C * H * W));

    int p_h = 0,p_w = 0, p_weight = 0;
    for (int i =0; i < weight_height; i++) {
      for (int j =0; j < weight_width; j++) {
        p_h = (i - weight_height/2) + h;
        p_w = (j - weight_width/2) + w;
        if (p_h >=0 && p_w >=0 && p_h < H && p_w < W) {
          p_weight = i * weight_width + j;
          int data_index = n*C*H*W + c*H*W + p_h*W + p_w;
          int weight_index = n*Size_Weight*H*w + p_weight*H*W + h*W + w;
          atomicAdd(grad_weight+weight_index, grad_top[i]*data[data_index]);
          atomicAdd(grad_data+data_index, grad_top[i]*weight[weight_index]);
          }
        }
      }
  }
}



int align_feature_cuda_forward_launcher(const at::Tensor data,
                                        const at::Tensor weight,
                                        const int weight_height,
                                        const int weight_width,
                                        const int N,
                                        const int C,
                                        const int Size_Weight,
                                        const int H,
                                        const int W,
                                        at::Tensor output) {
    AT_ASSERTM(data.dim() == 4, "data should be 4 dimensions");
    AT_ASSERTM(weight.dim() == 4, "weight should be 4 dimensions");
    AT_ASSERTM(output.dim() == 4, "output should be 4 dimensions");

    long size = N * C * H * W;
    dim3 grid(std::min(THCCeilDiv(size, 512L), 4096L));
    dim3 block(512);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        data.type(), "align feature forward", ([&] {
           align_feature_forward_kernel<scalar_t><<<grid, block>>>(
               size, data.data<scalar_t>(), weight.data<scalar_t>(), weight_height, weight_width,
               N, C, Size_Weight, H, W, output.data<scalar_t>());
        }));
    THCudaCheck(hipGetLastError());
    return 1;
}


int align_feature_cuda_backward_launcher(at::Tensor top_grad,
                                         at::Tensor data,
                                         at::Tensor weight,
                                         const int weight_height,
                                         const int weight_width,
                                         const int N,
                                         const int C,
                                         const int Size_Weight,
                                         const int H,
                                         const int W,
                                         at::Tensor grad_data,
                                         at::Tensor grad_weight) {
                                         
    AT_ASSERTM(data.dim() == 4, "data should be 4 dimensions");
    AT_ASSERTM(weight.dim() == 4, "weight should be 4 dimensions");
    AT_ASSERTM(top_grad.dim() == 4, "output should be 4 dimensions");
    AT_ASSERTM(grad_data.dim() == 4, "data shoud be 4 dimensions");
    AT_ASSERTM(grad_weight.dim() == 4, "weight should be 4 dimensions");

    long size = N * C * H * W;
    dim3 grid(std::min(THCCeilDiv(size, 512L), 4096L));
    dim3 block(512);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        data.type(), "align feature backward", ([&] {
           align_feature_backward_kernel<scalar_t><<<grid, block>>>(
               size, top_grad.data<scalar_t>(), data.data<scalar_t>(), weight.data<scalar_t>(), weight_height, weight_width,
               N, C, Size_Weight, H, W, grad_data.data<scalar_t>(), grad_weight.data<scalar_t>());
        }));
    return 1;
}
